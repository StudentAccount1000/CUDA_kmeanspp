#include "hip/hip_runtime.h"
#define MAX_ITERATIONS 50
// #define DATA_LENGTH 2000
#define DATA_LENGTH 140000
// #define DATA_LENGTH 283894
#define MAX_THREADS_PER_BLOCK 1024
#define K 7                         // K # of clusters
#define CUDA_FILEPATH_PREFIX "CUDA"

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <limits.h>
#include <stdlib.h>
#include <string>
#include <chrono>
#include <iostream>
#include <ctime>
#include <cstdlib>

#include "./include/Point.h"
#include "./include/Centroid.h"
#include "./include/IOFile.h"

#include <cfloat>

#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
#else
__device__ double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}
#endif

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#define CHECK_LAST_CUDA_ERROR() checkLast(__FILE__, __LINE__)
void checkLast(const char* const file, const int line)
{
    hipError_t const err{hipGetLastError()};
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << std::endl;
        // We don't exit when we encounter CUDA errors in this example.
        // std::exit(EXIT_FAILURE);
    }
}

// calculate minimum distance from datapoint to centroids
__global__ void calculate_min_distance(Point * datapoints, Centroid * centroids, int clusterID){
    int index = threadIdx.x + blockIdx.x * MAX_THREADS_PER_BLOCK;
    if (index < DATA_LENGTH) {
        // compute squares of x and  y
        double x_sqr = pow((datapoints[index].x - centroids[clusterID].x), 2.0);
        double y_sqr = pow((datapoints[index].y - centroids[clusterID].y), 2.0);

        // sum squares to get distance
        double distance = x_sqr + y_sqr;

        // update point cluster assignment if current distance is less than current minimum distance
        if (distance < datapoints[index].minDist) {
            datapoints[index].minDist = distance;
            datapoints[index].clusterID = clusterID;
        }
    }
}

// compute sums of datapoints within clusters
__global__ void compute_cluster_sums(Point * datapoints, Centroid * centroids, double * xsums, double * ysums) {
    int index = threadIdx.x + blockIdx.x * MAX_THREADS_PER_BLOCK;
    if (index < DATA_LENGTH) {
        // get current cluster ID
        int clusterID = datapoints[index].clusterID;

        // atomically add to avoid data race
        atomicAdd(&centroids[clusterID].nPoints, 1.0);
        atomicAdd(&xsums[clusterID], datapoints[index].x);
        atomicAdd(&ysums[clusterID], datapoints[index].y);

        // reset min distance
        datapoints[index].minDist = DBL_MAX;
    }
}

int main(int argc, char**argv) {
    int k = K;

    // check if user argument exists
    if (argc > 1) {
        k = atoi(argv[1]);
        if (k < 6 || k > 9) {
            printf("Please choose a K-value between 6-9\n");
            return 0;
        }
    }
    printf("K is: %d\n", k);
    
    // error checking
    if(k > DATA_LENGTH){
        printf("K must be less than the number of data points");
        return 1;
    }

    // allocate host memory
    Point * h_data = (Point*) malloc(sizeof(Point) * DATA_LENGTH); 
    Centroid * h_centroids = (Centroid*) malloc(sizeof(Centroid) * k);
    double * h_xsums = (double*) malloc(sizeof(double) * k);
    double * h_ysums = (double*) malloc(sizeof(double) * k);

    // allocate memory for GPU objects on host (later to be copied over to device)
    Point * d_data = (Point*) malloc(sizeof(Point) * DATA_LENGTH);
    Centroid * d_centroids = (Centroid*) malloc(sizeof(Centroid) * k);
    double * d_xsums = (double*) malloc(sizeof(double) * k);
    double * d_ysums = (double*) malloc(sizeof(double) * k);

    // read file data points to h_data
    read_file_to_arr(h_data);

    // use already computed centroid values as initial centroid values
    // see /test/init_centroids.py for details
    // const char* filepath = "";
    // switch (k)
    // {
    // case 6:
    //     filepath = "./data/half_centroids/k6_centroids.txt";
    //     break;
    // case 7:
    //     filepath = "./data/half_centroids/k7_centroids.txt";
    //     break;
    // case 8:
    //     filepath = "./data/half_centroids/k8_centroids.txt";
    //     break;
    // case 9:
    //     filepath = "./data/half_centroids/k9_centroids.txt";
    //     break;
    // default:
    //     filepath = "./data/half_centroids/k7_centroids.txt";
    //     break;
    // }

    // read centroids into centroids array
    // read_file_to_arr(h_centroids, filepath);

    // naive centroid selection – randomly select k centroids
    srand(time(NULL));
    for (int i = 0; i < k; ++i) {
        int randomIndex = rand() % DATA_LENGTH;
        std::cout << randomIndex << "\n";
        Centroid *c = new Centroid();
        c->x = latitude;
        c->y = longitude;
        c->id = curr_index;
        h_centroids[i] = *centroid;
    }

    // 512 threads and 4 blocks for current data
    int n_blocks = static_cast<int>(ceil(static_cast<double>(DATA_LENGTH)/static_cast<double>(MAX_THREADS_PER_BLOCK)));
    dim3 threads(MAX_THREADS_PER_BLOCK);
    dim3 blocks(n_blocks);

    // allocate memory on GPU and copy datapoints and centroids to GPU
    hipMalloc((void**) &d_data, sizeof(Point) * DATA_LENGTH);
    hipMalloc((void**) &d_centroids, sizeof(Centroid) * k);
    hipMalloc((void**) &d_xsums, sizeof(double) * k);
    hipMalloc((void**) &d_ysums, sizeof(double) * k);

    // check for cuda error
    CHECK_LAST_CUDA_ERROR();

    hipMemcpy(d_data, h_data, sizeof(Point) * DATA_LENGTH, hipMemcpyHostToDevice);
    hipMemcpy(d_centroids, h_centroids, sizeof(Centroid) * k, hipMemcpyHostToDevice);

    // check for cuda error
    CHECK_LAST_CUDA_ERROR();

    using std::chrono::high_resolution_clock;
    using std::chrono::duration_cast;
    using std::chrono::duration;
    using std::chrono::milliseconds;

    // start clock
    auto t1 = high_resolution_clock::now();

    int counter = 0;
    const double epsilon = 1e-6;

    // main loop
    while(counter < MAX_ITERATIONS) {
        // for each centroid, calculate the minimum distance to centroids
        for (int i = 0; i < k; i++) {
            calculate_min_distance<<<blocks, threads>>>(d_data, d_centroids, i);

            // make sure sums and centroids nPoints are reset
            // h_xsums[i] = 0;
            // h_ysums[i] = 0;
            // h_centroids[i].nPoints = 0;
        }

        // allocate memory on GPU for sums and copy resetted sums from host to device
        // hipMalloc((void**) &d_xsums, sizeof(double) * k);
        // hipMalloc((void**) &d_ysums, sizeof(double) * k);
        // hipMemcpy(d_xsums, h_xsums, sizeof(double) * k, hipMemcpyHostToDevice);
        // hipMemcpy(d_ysums, h_ysums, sizeof(double) * k, hipMemcpyHostToDevice);

        // for each cluster, compute the xsums and the ysums
        compute_cluster_sums<<<blocks, threads>>>(d_data, d_centroids, d_xsums, d_ysums);

        // copy centroids data and sums data from device to host
        hipMemcpy(h_centroids, d_centroids, sizeof(Centroid) * k, hipMemcpyDeviceToHost);
        hipMemcpy(h_xsums, d_xsums, sizeof(double) * k, hipMemcpyDeviceToHost);
        hipMemcpy(h_ysums, d_ysums, sizeof(double) * k, hipMemcpyDeviceToHost);

        // create a flag for convergence
        bool converges = false;

        // compute new centroids
        for (int i = 0; i < k; i++) {
            // copy old centroid coordinates
            double old_centroid_x = h_centroids[i].x;
            double old_centroid_y = h_centroids[i].y;

            // compute new centroids
            h_centroids[i].x = h_centroids[i].nPoints != 0 ? (h_xsums[i] / h_centroids[i].nPoints) : old_centroid_x;
            h_centroids[i].y = h_centroids[i].nPoints != 0 ? (h_ysums[i] / h_centroids[i].nPoints) : old_centroid_y;

            // reset sums and cluster data points count
            // h_xsums[i] = 0;
            // h_ysums[i] = 0;
            // h_centroids[i].nPoints = 0;

            // compare new centroids with old_centroids
            // if new centroids are same as old_centroids, set converge to true
            if (std::abs(h_centroids[i].x - old_centroid_x) < epsilon &&
                std::abs(h_centroids[i].y - old_centroid_y) < epsilon) {
                converges = true;
            } else {
		converges = false;
	    }
        }

        // if converge, break loop
        if (converges) {
            printf("converges at iteration: %d\n", counter+1);
            break;
        }

	// copy newly computed centroids and resetted sums from host to device
        hipMemcpy(d_centroids, h_centroids, sizeof(Centroid) * k, hipMemcpyHostToDevice);
        // hipMemcpy(d_xsums, h_xsums, sizeof(double) * k, hipMemcpyHostToDevice);
        // hipMemcpy(d_ysums, h_ysums, sizeof(double) * k, hipMemcpyHostToDevice);

        // deallocate and reset sums
        // hipFree(d_xsums);
        // hipFree(d_ysums);

        counter++;
    }

    // end clock
    auto t2 = high_resolution_clock::now();

    // // compute time took to execute algorithm
    duration<double, std::milli> ms_double = t2 - t1;

    std::cout << ms_double.count() << "ms\n";

    // copy datapoints with clusterID data from device to host
    hipMemcpy(h_data, d_data, sizeof(Point) * DATA_LENGTH, hipMemcpyDeviceToHost);
    hipMemcpy(h_centroids, d_centroids, sizeof(Centroid) * k, hipMemcpyDeviceToHost);

    // // deallocate device memory
    hipFree(d_data);
    hipFree(d_centroids);
    hipFree(d_xsums);
    hipFree(d_ysums);
    printf("freed device  memory\n");

    // deallocate host memory
    // free(h_data);
    // free(h_centroids);
    // free(h_xsums);
    // free(h_ysums);
    // free(d_data);
    // free(d_centroids);
    // free(d_xsums);
    // free(d_ysums);
    // printf("freed host mem\n");

    const std::string filepath_prefix = CUDA_FILEPATH_PREFIX;

    write_array_to_csv(filepath_prefix, h_data, k);
    write_centroids_to_txt(filepath_prefix, h_centroids, k);

    printf("finished writing result to file\n");

    return 0;
}
